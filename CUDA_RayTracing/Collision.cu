#include "hip/hip_runtime.h"
#include "Primitive.cuh"

__device__ void CylinderIntersect(Cuda_Primitive* primitive, const double3* origin, const double3* direction, Cuda_Collision* collision)
{
    double3 V = normalize(*direction);
    // Body
    {
        // Cylinder collision detection
        double3 d = primitive->data.cylinder.O2 - primitive->data.cylinder.O1;
        double3 m = *origin - primitive->data.cylinder.O1;
        double3 n = *origin - primitive->data.cylinder.O2;
        double md = dot(m, d);
        double nd = dot(n, d);
        double dd = dot(d, d);
        double r2 = primitive->data.cylinder.R * primitive->data.cylinder.R;

        // Cylinder body collision detection
        double3 dNorm = normalize(d);
        //double3 mProj = m - dot(m, dNorm) * dNorm;
        //double3 VProj = V - dot(V, dNorm) * dNorm;
        double3 mProj = cross(m, dNorm);
        double3 VProj = cross(V, dNorm);
        double a = dot(VProj, VProj);
        double b = 2.0 * dot(VProj, mProj);
        double c = dot(mProj, mProj) - r2;
        double det = b * b - 4 * a * c;

        if (det >= 0)
        {
            det = sqrt(det);
            double t0 = (-b - det) / (2.0 * a);
            double t1 = (-b + det) / (2.0 * a);

            double t = fmin(t0, t1);
            if (t < 0) t = fmax(t0, t1);
            if (t >= 0)
            {
                double3 P = *origin + V * t;
                //double3 Q = P - primitive->data.cylinder.O1;
                double u = dot(P - primitive->data.cylinder.O1, dNorm);

                if (u >= 0 && u <= length(d))
                {
                    // Indicate body
                    collision->dist = t;
                    collision->C = P;
                    collision->isCollide = true;
                    collision->collide_primitive = primitive;
                    collision->front = false;
                    collision->N = normalize(P - (primitive->data.cylinder.O1 + dNorm * u));
                    return;
                }
                else
                {
                }
            }
        }
    }
    // Top Cap
    {
        double3 N = normalize(primitive->data.cylinder.O1 - primitive->data.cylinder.O2);
        double denom = dot(N, V);
        if (fabs(denom) >= 1e-6)
        {
            float t = dot(primitive->data.cylinder.O2 - *origin, N) / denom;
            if (t >= 1e-6)
            {
                double3 P = *origin + V * t;
                double3 Q = P - primitive->data.cylinder.O2;
                if (dot(Q, Q) <= primitive->data.cylinder.R * primitive->data.cylinder.R)
                {
                    collision->dist = t;
                    collision->C = P;
                    collision->front = true;
                    collision->N = (denom < 0) ? N : -N;
                    collision->isCollide = true;
                    collision->collide_primitive = primitive;
                    return;
                }
            }
        }
    }
    // Bottom cap
    {
        double3 N = normalize(primitive->data.cylinder.O2 - primitive->data.cylinder.O1);
        double denom = dot(N, V);
        if (fabs(denom) >= 1e-6)
        {
            float t = dot(primitive->data.cylinder.O1 - *origin, N) / denom;
            if (t >= 1e-6)
            {
                double3 P = *origin + V * t;
                double3 Q = P - primitive->data.cylinder.O1;
                if (dot(Q, Q) <= primitive->data.cylinder.R * primitive->data.cylinder.R)
                {
                    collision->dist = t;
                    collision->C = P;
                    collision->front = true;
                    collision->N = (denom < 0) ? N : -N;
                    collision->isCollide = true;
                    collision->collide_primitive = primitive;
                    return;
                }
            }
        }
    }
}

// Function to check triangle-ray intersection using M�ller�Trumbore algorithm
__device__ bool TriangleIntersect(const Cuda_Triangle* triangle, const double3* origin, const double3* direction, double* t, double3* P, double3* N)
{
    const double EPSILON = 1e-6;
    double3 V1 = triangle->O1;
    double3 V2 = triangle->O2;
    double3 V3 = triangle->O3;

    double3 edge1 = V2 - V1;
    double3 edge2 = V3 - V1;
    double3 h = cross(*direction, edge2);
    double a = dot(edge1, h);

    if (fabs(a) < EPSILON)
        return false; // This ray is parallel to this triangle.

    double f = 1.0 / a;
    double3 s = *origin - V1;
    double u = f * dot(s, h);

    if (u < 0.0 || u > 1.0)
        return false;

    double3 q = cross(s, edge1);
    double v = f * dot(*direction, q);

    if (v < 0.0 || u + v > 1.0)
        return false;

    double temp = f * dot(edge2, q);

    if (temp > EPSILON) // ray intersection
    {
        *t = temp;
        *P = *origin + (*direction) * (*t);
        *N = normalize(cross(edge1, edge2));
        return true;
    }
    else // This means that there is a line intersection but not a ray intersection.
        return false;
}

__device__ bool intersect(Cuda_Primitive* primitive, const double3 * origin, const double3 * direction, Cuda_Collision* collision)
{
    switch (primitive->type)
    {
        case Cuda_Primitive_Type_Sphere:
        {
            double3 V = normalize(*direction);
            double3 P = *origin - primitive->data.sphere.O;
            double b = -dot(P, V);
            double det = b * b - dot(P, P) + primitive->data.sphere.R * primitive->data.sphere.R;

            if (det > 1e-6)
            {
                det = sqrt(det);
                double x1 = b - det, x2 = b + det;
                if (x2 < 1e-6) return false;
                collision->front = (x1 > 1e-6);
                collision->dist = collision->front ? x1 : x2;
                collision->C = *origin + (V * collision->dist);
                collision->N = normalize(collision->C - primitive->data.sphere.O);
                if (collision->front == false) collision->N = -collision->N;
                collision->isCollide = true;
                collision->collide_primitive = primitive;
            }
            break;
        }
        case Cuda_Primitive_Type_Plane:
        {
            double3 V = normalize(*direction);
            double3 N = normalize(primitive->data.plane.N);
            double denom = dot(N, V);
            if (fabs(denom) >= 1e-6)
            {
                float t = dot(N * primitive->data.plane.R - *origin, N) / denom;
                if (t >= 1e-6)
                {
                    collision->dist = t;
                    collision->C = *origin + V * collision->dist;
                    collision->front = (denom < 0);
                    collision->N = collision->front ? N : -N;
                    collision->isCollide = true;
                    collision->collide_primitive = primitive;
                }
            }
            break;
        }
        case Cuda_Primitive_Type_Square:
        {

            //ray_V = ray_V.GetUnitVector();
            //auto N = (Dx * Dy).GetUnitVector();
            //double d = N.Dot(ray_V);

            //if (fabs(d) < EPS) {
            //    return ret;
            //}

            //// solve equation
            //double t = (O - ray_O).Dot(N) / d;
            //if (t < EPS) {
            //    return ret;
            //}
            //auto P = ray_O + ray_V * t;

            //// check whether inside square
            //double DxLen2 = Dx.Module2();
            //double DyLen2 = Dy.Module2();

            //double x2 = abs((P - O).Dot(Dx));
            //double y2 = abs((P - O).Dot(Dy));
            //if (x2 > DxLen2 || y2 > DyLen2) {
            //    return ret;
            //}

            //ret.dist = t;
            //ret.front = (d < 0);
            //ret.O3 = P;
            //ret.N = (ret.front) ? N : -N;
            //ret.isCollide = true;
            //ret.collide_primitive = this;
            //return ret;
            
            double3 V = normalize(*direction);
            double3 N = normalize(cross(primitive->data.square.Dx, primitive->data.square.Dy));
            double denom = dot(N, V);
            if (fabs(denom) < 1e-6) return false;
            
            float t = dot(primitive->data.square.O - *origin, N) / denom;
            
            if (t < 1e-6) return false;

            double3 P = *origin + V * t;
            double DxLen2 = dot(primitive->data.square.Dx, primitive->data.square.Dx);
            double DyLen2 = dot(primitive->data.square.Dy, primitive->data.square.Dy);
            double x2 = fabs(dot(P - primitive->data.square.O, primitive->data.square.Dx));
            double y2 = fabs(dot(P - primitive->data.square.O, primitive->data.square.Dy));
            if (x2 > DxLen2 || y2 > DyLen2) return false;

            collision->dist = t;
            collision->front = (denom < 0);
            collision->C = P;
            collision->N = collision->front ? N : -N;
            collision->isCollide = true;
            collision->collide_primitive = primitive;
            break;
        }
        case Cuda_Primitive_Type_Cylinder:
        {
            CylinderIntersect(primitive, origin, direction, collision);
            break;
        }
        case Cuda_Primitive_Type_Bezier:
        {
            BezierIntersect(primitive, origin, direction, collision);
            break;
        }
        case Cuda_Primitive_Type_Triangle:
        {
            double t;
            double3 P, N;
            if (TriangleIntersect(&(primitive->data.triangle), origin, direction, &t, &P, &N))
            {
                collision->dist = t;
                collision->C = P;
                collision->N = N;
                collision->isCollide = true;
                collision->collide_primitive = primitive;
                collision->front = dot(*direction, N) < 0;
                return true;
            }
            break;
        }
        case Cuda_Primitive_Type_Mesh:
        {
            MeshIntersect(primitive, origin, direction, collision);
            break;
        }
    }

    return collision->isCollide;
}
