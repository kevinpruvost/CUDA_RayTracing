#include "hip/hip_runtime.h"
#include "Primitive.cuh"

__device__ void CylinderIntersect(Cuda_Primitive* primitive, const double3* origin, const double3* direction, Cuda_Collision* collision)
{
    double3 V = normalize(*direction);
    // Body
    {
        // Cylinder collision detection
        double3 d = primitive->data.cylinder.O2 - primitive->data.cylinder.O1;
        double3 m = *origin - primitive->data.cylinder.O1;
        double3 n = *origin - primitive->data.cylinder.O2;
        double md = dot(m, d);
        double nd = dot(n, d);
        double dd = dot(d, d);
        double nn = dot(n, n);
        double mn = dot(m, n);
        double a = dd - md * md / dd;
        double b = dd - nd * nd / dd;
        double c = dd - mn * mn / dd;
        double e = dd - nn * nn / dd;
        double f = dd - md * nd / dd;
        double r2 = primitive->data.cylinder.R * primitive->data.cylinder.R;

        // Test for collision with infinite cylinder
        double a1 = dot(V, V) - pow(dot(V, d), 2) / dd;
        double b1 = dot(m, V) - md * dot(V, d) / dd;
        double c1 = dot(m, m) - pow(md, 2) / dd - r2;
        double det = b1 * b1 - a1 * c1;
        if (det >= 0)
        {
            det = sqrt(det);
            double t0 = (-b1 - det) / a1;
            double t1 = (-b1 + det) / a1;

            // Check for collision within cylinder caps
            if (t1 >= 1e-6)
            {
                if (t0 < 1e-6) t0 = t1;

                double3 P = *origin + V * t0;
                double3 Q = (P - primitive->data.cylinder.O1);
                double u = dot(Q, d) / dd;
                if (!(u < 0 || u > 1))
                {
                    collision->dist = t0;
                    collision->front = (dot(V, d) >= 0);
                    collision->C = P;
                    collision->N = normalize(P - (primitive->data.cylinder.O1 + u * d));
                    if (!collision->front) collision->N = -collision->N;
                    collision->isCollide = true;
                    collision->collide_primitive = primitive;
                    return;
                }
            }
        }
    }
    // Top Cap
    {
        double3 N = normalize(primitive->data.cylinder.O1 - primitive->data.cylinder.O2);
        double denom = dot(N, V);
        if (fabs(denom) >= 1e-6)
        {
            float t = dot(primitive->data.cylinder.O2 - *origin, N) / denom;
            if (t >= 1e-6)
            {
                double3 P = *origin + V * t;
                double3 Q = P - primitive->data.cylinder.O2;
                if (dot(Q, Q) <= primitive->data.cylinder.R * primitive->data.cylinder.R)
                {
                    collision->dist = t;
                    collision->C = P;
                    collision->front = (denom < 0);
                    collision->N = collision->front ? -N : N;
                    collision->isCollide = true;
                    collision->collide_primitive = primitive;
                    return;
                }
            }
        }
    }
    // Bottom cap
    {
        double3 N = normalize(primitive->data.cylinder.O2 - primitive->data.cylinder.O1);
        double denom = dot(N, V);
        if (fabs(denom) >= 1e-6)
        {
            float t = dot(primitive->data.cylinder.O1 - *origin, N) / denom;
            if (t >= 1e-6)
            {
                double3 P = *origin + V * t;
                double3 Q = P - primitive->data.cylinder.O1;
                if (dot(Q, Q) <= primitive->data.cylinder.R * primitive->data.cylinder.R)
                {
                    collision->dist = t;
                    collision->C = P;
                    collision->front = (denom < 0);
                    collision->N = collision->front ? -N : N;
                    collision->isCollide = true;
                    collision->collide_primitive = primitive;
                    return;
                }
            }
        }
    }
}

__device__ bool intersect(Cuda_Primitive* primitive, const double3 * origin, const double3 * direction, Cuda_Collision* collision)
{
    switch (primitive->type)
    {
        case Cuda_Primitive_Type_Sphere:
        {
            double3 V = normalize(*direction);
            double3 P = *origin - primitive->data.sphere.O;
            double b = -dot(P, V);
            double det = b * b - dot(P, P) + primitive->data.sphere.R * primitive->data.sphere.R;

            if (det > 1e-6)
            {
                det = sqrt(det);
                double x1 = b - det, x2 = b + det;
                if (x2 < 1e-6) return false;
                collision->front = (x1 > 1e-6);
                collision->dist = collision->front ? x1 : x2;
                collision->C = *origin + (V * collision->dist);
                collision->N = normalize(collision->C - primitive->data.sphere.O);
                if (collision->front == false) collision->N = -collision->N;
                collision->isCollide = true;
                collision->collide_primitive = primitive;
            }
            break;
        }
        case Cuda_Primitive_Type_Plane:
        {
            double3 V = normalize(*direction);
            double3 N = normalize(primitive->data.plane.N);
            double denom = dot(N, V);
            if (fabs(denom) >= 1e-6)
            {
                float t = dot(N * primitive->data.plane.R - *origin, N) / denom;
                if (t >= 1e-6)
                {
                    collision->dist = t;
                    collision->C = *origin + V * collision->dist;
                    collision->front = (denom < 0);
                    collision->N = collision->front ? N : -N;
                    collision->isCollide = true;
                    collision->collide_primitive = primitive;
                }
            }
            break;
        }
        case Cuda_Primitive_Type_Square:
        {

            //ray_V = ray_V.GetUnitVector();
            //auto N = (Dx * Dy).GetUnitVector();
            //double d = N.Dot(ray_V);

            //if (fabs(d) < EPS) {
            //    return ret;
            //}

            //// solve equation
            //double t = (O - ray_O).Dot(N) / d;
            //if (t < EPS) {
            //    return ret;
            //}
            //auto P = ray_O + ray_V * t;

            //// check whether inside square
            //double DxLen2 = Dx.Module2();
            //double DyLen2 = Dy.Module2();

            //double x2 = abs((P - O).Dot(Dx));
            //double y2 = abs((P - O).Dot(Dy));
            //if (x2 > DxLen2 || y2 > DyLen2) {
            //    return ret;
            //}

            //ret.dist = t;
            //ret.front = (d < 0);
            //ret.C = P;
            //ret.N = (ret.front) ? N : -N;
            //ret.isCollide = true;
            //ret.collide_primitive = this;
            //return ret;
            
            double3 V = normalize(*direction);
            double3 N = normalize(cross(primitive->data.square.Dx, primitive->data.square.Dy));
            double denom = dot(N, V);
            if (fabs(denom) < 1e-6) return false;
            
            float t = dot(primitive->data.square.O - *origin, N) / denom;
            
            if (t < 1e-6) return false;

            double3 P = *origin + V * t;
            double DxLen2 = dot(primitive->data.square.Dx, primitive->data.square.Dx);
            double DyLen2 = dot(primitive->data.square.Dy, primitive->data.square.Dy);
            double x2 = fabs(dot(P - primitive->data.square.O, primitive->data.square.Dx));
            double y2 = fabs(dot(P - primitive->data.square.O, primitive->data.square.Dy));
            if (x2 > DxLen2 || y2 > DyLen2) return false;

            collision->dist = t;
            collision->front = (denom < 0);
            collision->C = P;
            collision->N = collision->front ? N : -N;
            collision->isCollide = true;
            collision->collide_primitive = primitive;
            break;
        }
        case Cuda_Primitive_Type_Cylinder:
        {
            CylinderIntersect(primitive, origin, direction, collision);
            break;
        }
        case Cuda_Primitive_Type_Bezier:
        {
            // Bezier collision detection (simple approximation for demonstration)
            double3 V = normalize(*direction);
            double3 P = *origin;
            double t_min = 1e-6;
            double t_max = 1e10;

            // Iterate over the control points (simple approximation, you may need a more complex approach)
            for (int i = 0; i < primitive->data.bezier.degree; ++i) {
                double3 B = lerp(primitive->data.bezier.O1, primitive->data.bezier.O2, i / (double)primitive->data.bezier.degree);
                double r = primitive->data.bezier.R[i];
                double3 m = P - B;
                double b = dot(m, V);
                double c = dot(m, m) - r * r;
                if (c > 0.0 && b > 0.0) continue;

                double disc = b * b - c;
                if (disc < 0.0) continue;

                double t0 = -b - sqrt(disc);
                if (t0 < t_min || t0 > t_max) continue;

                collision->dist = t0;
                collision->C = P + V * t0;
                collision->N = normalize(collision->C - B);
                collision->front = (dot(V, collision->N) < 0);
                if (!collision->front) collision->N = -collision->N;
                collision->isCollide = true;
                collision->collide_primitive = primitive;
                break;
            }
            break;
        }
    }

    return collision->isCollide;
}
