#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math_functions.h>
#include <>
#include <iostream>
#include "RayTracer.cuh"

//// Utility function to normalize a float3
//__device__ float3 normalize(const float3& v) {
//    float len = rsqrtf(v.x * v.x + v.y * v.y + v.z * v.z);
//    return {
//        v.x / len,
//        v.y / len,
//        v.z / len
//    };
//}
//
//// Utility function to perform linear interpolation
//__device__ float3 lerp(const float3& a, const float3& b, float t) {
//    return {
//        a.x + t * (b.x - a.x),
//        a.y + t * (b.y - a.y),
//        a.z + t * (b.z - a.z)
//    };
//}
//
//// Simple background color function
//__device__ float3 traceRay(const float3& origin, const float3& direction) {
//    // Linearly interpolate between white and blue based on the y coordinate
//    float t = 0.5f * (direction.y + 1.0f);
//    return lerp({ 1.0f, 1.0f, 1.0f }, { 0.5f, 0.7f, 1.0f }, t);
//}

__global__ void rayTraceKernel(hipSurfaceObject_t surface, int width, int height, Cuda_Scene* scene) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    double u = float(x) / float(width);
    double v = float(y) / float(height);

    //float3 origin = make_float3(0.0f, 0.0f, 0.0f);
    //float3 direction = normalize(make_float3(u - 0.5f, v - 0.5f, -1.0f));

    float3 color = traceRay(scene, u, v);
    uchar4 outputColor;
    outputColor = make_uchar4(color.x * 255, 0, 0, 255);
    //for (int i = 0; i < scene->primitiveCount; ++i) {
    //    if (scene->primitives[i].material.texture_width != 0) {
    //        uchar3* texture = scene->primitives[i].material.texture;
    //        int texture_x = static_cast<int>(u * scene->primitives[i].material.texture_width);
    //        int texture_y = static_cast<int>(v * scene->primitives[i].material.texture_height);
    //        outputColor = make_uchar4(
    //            texture[texture_x * scene->primitives[i].material.texture_height + texture_y].x,
    //            texture[texture_x * scene->primitives[i].material.texture_height + texture_y].y,
    //            texture[texture_x * scene->primitives[i].material.texture_height + texture_y].z,
    //            255);
    //    }
    //}

    surf2Dwrite(outputColor, surface, x * sizeof(uchar4), y);
}

#ifdef _DEBUG
__global__ void debugTest(Cuda_Scene* scene) {
    // Printing Camera information
    //printf("Camera Information:\n");
    //printf("Position: (%f, %f, %f)\n", scene->camera.O.x, scene->camera.O.y, scene->camera.O.z);
    //printf("Direction: (%f, %f, %f)\n", scene->camera.N.x, scene->camera.N.y, scene->camera.N.z);

    //// Printing Light information
    //printf("Light Information:\n");
    //for (int i = 0; i < scene->lightCount; i++) {
    //    printf("Light %d:\n", i);
    //    printf("Position: (%f, %f, %f)\n", scene->lights[i].O.x, scene->lights[i].O.y, scene->lights[i].O.z);
    //    printf("Color: (%f, %f, %f)\n", scene->lights[i].color.x, scene->lights[i].color.y, scene->lights[i].color.z);
    //    printf("Type: %d\n", scene->lights[i].type);
    //}
    for (int i = 0; i < scene->primitiveCount; ++i) {
        if (scene->primitives[i].material.texture_width != 0) {
            uchar3 * texture = scene->primitives[i].material.texture;
            printf("Test read: Width:%d | height:%d | texture: %d,%d,%d\n", scene->primitives[i].material.texture_width, scene->primitives[i].material.texture_height, texture[0].x, texture[0].y, texture[0].z);
        }
    }
}
#endif

// Wrapper function to launch the kernel
void launchRayTraceKernel(hipSurfaceObject_t surface, int width, int height, Cuda_Scene * scene)
{
    // Define block and grid sizes
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((width + threadsPerBlock.x - 1) / threadsPerBlock.x, (height + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Launch the kernel
#ifdef _DEBUG
    //debugTest << <1, 1 >> > (scene);
#endif
    rayTraceKernel<<<blocksPerGrid, threadsPerBlock >> > (surface, width, height, scene);

    // Ensure kernel launch is successful
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch rayTraceKernel (error code %s)!\n", hipGetErrorString(err));
    }

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();
}