#include <hip/hip_runtime.h>
#include <math_functions.h>

#include <iostream>

// Utility function to normalize a float3
__device__ float3 normalize(const float3& v) {
    float len = rsqrtf(v.x * v.x + v.y * v.y + v.z * v.z);
    return {
        v.x / len,
        v.y / len,
        v.z / len
    };
}

// Utility function to perform linear interpolation
__device__ float3 lerp(const float3& a, const float3& b, float t) {
    return {
        a.x + t * (b.x - a.x),
        a.y + t * (b.y - a.y),
        a.z + t * (b.z - a.z)
    };
}

// Simple background color function
__device__ float3 traceRay(const float3& origin, const float3& direction) {
    // Linearly interpolate between white and blue based on the y coordinate
    float t = 0.5f * (direction.y + 1.0f);
    return lerp({ 1.0f, 1.0f, 1.0f }, { 0.5f, 0.7f, 1.0f }, t);
}

__global__ void rayTraceKernel(uchar4* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    int idx = y * width + x;

    float u = float(x) / float(width);
    float v = float(y) / float(height);

    float3 origin = make_float3(0.0f, 0.0f, 0.0f);
    float3 direction = normalize(make_float3(u - 0.5f, v - 0.5f, -1.0f));

    float3 color = traceRay(origin, direction);

    output[idx] = make_uchar4(color.x * 255, color.y * 255, color.z * 255, 255);
}

// Wrapper function to launch the kernel
void launchRayTraceKernel(uchar4 * d_output, int width, int height) {
    // Define block and grid sizes
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    // Launch the kernel
    rayTraceKernel<<<gridSize, blockSize>>>(d_output, width, height);

    // Ensure kernel launch is successful
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch rayTraceKernel (error code %s)!\n", hipGetErrorString(err));
    }

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();
}